#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include <assert.h>


#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// Lab4: You can use any other block size you wish.
#define BLOCK_SIZE 256

// Lab4: Host Helper Functions (allocate your own data structure...)

__global__ void scan(float *g_odata, float *g_idata, const int n);
void prescanArray(float *outArray, float *inArray, int numElements);

// Lab4: Device Functions


// Lab4: Kernel Functions
// n: block size. one thread can handle two elements
__global__ void scan(float *g_odata, float *g_idata, const int n)
{





 	extern __shared__ float temp[]; // allocated on invocation: only needs to be as big as num threads in block

	int thid = threadIdx.x;	//thread id in block
	int gid = blockIdx.x*blockDim.x + thid;	//global id
	
	int offset = 1;


	
	//Loop all elements partitioned to a block in input array
		



	
	//Every thread handles two elements
	temp[2*thid] = g_idata[2*thid]; // load input into shared memory
        temp[2*thid+1] = g_idata[2*thid+1];

	// load input into shared memory.
 	// This is exclusive scan, so shift right by one and set first element to 0

	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree
 	{
 		__syncthreads();
 		if (thid < d)
 		{
			int ai = offset*(2*thid+1)-1;
 			int bi = offset*(2*thid+2)-1;
 			temp[bi] += temp[ai];
 		}
 		offset *= 2;
 	}
 	if (thid == 0) { temp[n - 1] = 0; } // clear the last element
 	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
 	{
 		offset >>= 1;
 		__syncthreads();
		 if (thid < d)
 		{
			int ai = offset*(2*thid+1)-1;
 			int bi = offset*(2*thid+2)-1;
 			float t = temp[ai];
 			temp[ai] = temp[bi];
 			temp[bi] += t;
 		}
 	}
 	__syncthreads();
 	g_odata[2*thid] = temp[2*thid]; // write results to device memory
 	g_odata[2*thid+1] = temp[2*thid+1];


}


	
	

// **===-------- Lab4: Modify the body of this function -----------===**
// You may need to make multiple kernel calls, make your own kernel
// function in this file, and then call them from here.
void prescanArray(float *outArray, float *inArray, int numElements)
{
	// Divide input array into blocks
	// Remember that each thread can handle two elements
	// BLOCK_SIZE is set above as a constnt
//	int numBlocks = ceil(numElements/BLOCK_SIZE);
	
	// Allocate global device memory for arrays to communicate sum data
//	cudaMalloc(


//	int nepb = num_elements/gridDim.x; //assuming input array is a power of 2 already
	

	dim3 dimGrid(1);
	dim3 dimBlock(BLOCK_SIZE);
	
	//BEV: added all below this point
	//dim3 dimGrid(numBlocks);
	//dim3 dimBlock(BLOCK_SIZE);
    	scan<<<dimGrid, dimBlock, 2*sizeof(float)*numElements+1>>>(outArray, inArray, numElements);
	//scan<<<dimGrid, dimBlock, 2*sizeof(float)*numElements+1>>>(outArray, inArray, BLOCK_SIZE); 
    
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
